#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <stdio.h>
#include <iostream>

// Constant buffers
// TODO Aufgabe 3)
__constant__ float camera_Buffer[8];
__constant__ float lightdat_Buffer[12];

// Normal buffers
float4* result_Buffer;
int2 resinfo_Buffer;
float4* spheres_Buffer;
int4* faces_Buffer;
float4* vertices_Buffer;
float4* normals_Buffer;
int* counts_Buffer;
float* materials_Buffer;
int* matidx_Buffer;
float4* objbounds_Buffer;
int* objmeshes_Buffer;
int4* result_AnaVals;

hipError_t setupCuda() {
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	return cudaStatus;
}

////////// END BLOCK: DO NOT TOUCH ANY CODE IN THIS HELPER FUNCTIONS

/*
* width = Horizontal no. of pixels of rendered image
* height = Vertical no. of pixels of rendered image
*/
hipError_t setupCudaMemory(SplitScene* scene, 
	const int width, const int height, const int facecnt,
	float* spheres, int* faces, float* vertices, float* normals,
	int* loc_counts, float* materials, int* matidx,
	float* objbounds, int* objmeshes)
{
	hipError_t cudaStatus;
	size_t size;

	// Allocate and fill GPU buffers (faces, vertices, result, ...)    .
	// Buffer for result picture
	// TODO Aufgabe 1)
	size = width*height;
    cudaStatus= hipMalloc((void**)&result_Buffer,size* sizeof(float4));
    if (cudaStatus != hipSuccess) {
        const char *const err_str = hipGetErrorString(cudaStatus);
        std::cerr<<"result_Buffer memory allocation failed!"<<std::endl;
        std::cerr << "Cuda error in " << __FILE__ << ":" << __LINE__ - 4
                  << ": " << err_str << " (" << cudaStatus << ")" << std::endl;
		return cudaStatus;
    }
    hipMemset(result_Buffer, 0, size* sizeof(float)*4);
    if (cudaStatus != hipSuccess) {
        const char *const err_str = hipGetErrorString(cudaStatus);
        std::cerr<<"result_Buffer memory initialization failed!"<<std::endl;
        std::cerr << "Cuda error in " << __FILE__ << ":" << __LINE__ - 4
                  << ": " << err_str << " (" << cudaStatus << ")" << std::endl;
		return cudaStatus;
    }


    // Buffer for vertices

    // TODO Aufgabe 2)
	size = scene->vtxcnt;
    hipMalloc((void**)&vertices_Buffer, sizeof(float4)*size);
    if (cudaStatus != hipSuccess) {
        const char *const err_str = hipGetErrorString(cudaStatus);
        std::cerr<<"vertices_Buffer memory allocation failed!"<<std::endl;
        std::cerr << "Cuda error in " << __FILE__ << ":" << __LINE__ - 4
                  << ": " << err_str << " (" << cudaStatus << ")" << std::endl;
		return cudaStatus;
    }
    hipMemcpy(vertices_Buffer,vertices,sizeof(float)*4*size,hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        const char *const err_str = hipGetErrorString(cudaStatus);
        std::cerr<<"vertices  memory copying failed!"<<std::endl;
        std::cerr << "Cuda error in " << __FILE__ << ":" << __LINE__ - 4
                  << ": " << err_str << " (" << cudaStatus << ")" << std::endl;
		return cudaStatus;
    }


	// Buffer for camera
	float camera[8];
	camera[0] = scene->cam.pos->point2; // x
	camera[1] = scene->cam.pos->point3; // y
	camera[2] = scene->cam.pos->point4; // z
	camera[3] = scene->cam.cameraFOVwidth; // cameraFOVwidth
	camera[4] = scene->cam.cameraFOVheight; // cameraFOVheight
	camera[5] = scene->cam.pixelWidth; // pixelWidth
	camera[6] = scene->cam.pixelHeight; //camera.s[4] / height; // pixelHeight
	camera[7] = 0.0f; // dummy

	// Copy camera to constant camera_Buffer

    // TODO Aufgabe 3)
    hipMemcpyToSymbol(HIP_SYMBOL(camera_Buffer), camera,8* sizeof(float));
    if (cudaStatus != hipSuccess) {
        const char *const err_str = hipGetErrorString(cudaStatus);
        std::cerr<<"camera_Buffer memory allocation failed!"<<std::endl;
        std::cerr << "Cuda error in " << __FILE__ << ":" << __LINE__ - 4
                  << ": " << err_str << " (" << cudaStatus << ")" << std::endl;
		return cudaStatus;
    }




	// Buffer for light
	float lightdat[12];
	lightdat[0] = scene->light.pos.point2;
	lightdat[1] = scene->light.pos.point3;
	lightdat[2] = scene->light.pos.point4;
	lightdat[3] = scene->light.ambient->r;
	lightdat[4] = scene->light.ambient->g;
	lightdat[5] = scene->light.ambient->b;
	lightdat[6] = scene->light.diffuse->r;
	lightdat[7] = scene->light.diffuse->g;
	lightdat[8] = scene->light.diffuse->b;
	lightdat[9] = scene->light.specular->r;
	lightdat[10] = scene->light.specular->g;
	lightdat[11] = scene->light.specular->b;

	// Copy light to constant light_Buffer

    // TODO Aufgabe 3)
    hipMemcpyToSymbol(HIP_SYMBOL(lightdat_Buffer), lightdat, 12* sizeof(float));
    if (cudaStatus != hipSuccess) {
        const char *const err_str = hipGetErrorString(cudaStatus);
        std::cerr<<"lightdat_Buffer memory allocation failed!"<<std::endl;
        std::cerr << "Cuda error in " << __FILE__ << ":" << __LINE__ - 4
                  << ": " << err_str << " (" << cudaStatus << ")" << std::endl;
		return cudaStatus;
    }




	/////// DO NOT TOUCH CODE OF THIS FUNCTION FROM HERE ON! ///////
	// Buffer for resolution info
	resinfo_Buffer.x = width;
	resinfo_Buffer.y = height;

	// Buffer for bounding spheres
	size = scene->getMeshCnt() ;
	cudaStatus = hipMalloc((void**)&spheres_Buffer, size * sizeof(float4));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for spheres_Buffer failed!");
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(spheres_Buffer, spheres, size * sizeof(float) * 4, 
		hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for spheres_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for faces
	size = facecnt;
	cudaStatus = hipMalloc((void**)&faces_Buffer, size * sizeof(int4));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for faces_Buffer failed!");
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(faces_Buffer, faces, size * sizeof(int) * 4,
		hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for faces_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for normals
	size = scene->vtxcnt;
	cudaStatus = hipMalloc((void**)&normals_Buffer, size * sizeof(float4));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for normals_Buffer failed!");
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(normals_Buffer, normals, size * sizeof(float) * 4,
		hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for normals_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for counts
	size = scene->getMeshCnt();
	cudaStatus = hipMalloc((void**)&counts_Buffer, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for counts_Buffer failed!");
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(counts_Buffer, loc_counts, size * sizeof(int),
		hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for counts_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for materials
	size = scene->getMaterialCount() * 16;
	cudaStatus = hipMalloc((void**)&materials_Buffer, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for materials_Buffer failed!");
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(materials_Buffer, materials, size * sizeof(float),
		hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for materials_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for material indices
	size = scene->getMeshCnt();
	cudaStatus = hipMalloc((void**)&matidx_Buffer, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for matidx_Buffer failed!");
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(matidx_Buffer, matidx, size * sizeof(int),
		hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for matidx_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for object bounds
	size = scene->objcnt;
	cudaStatus = hipMalloc((void**)&objbounds_Buffer, size * sizeof(float4));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for objbounds_Buffer failed!");
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(objbounds_Buffer, objbounds, size * sizeof(float)*4,
		hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for objbounds_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for object meshes
	size = scene->objcnt;
	cudaStatus = hipMalloc((void**)&objmeshes_Buffer, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for objmeshes_Buffer failed!");
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(objmeshes_Buffer, objmeshes, size * sizeof(int),
		hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for objmeshes_Buffer failed!");
		return cudaStatus;
	}

	return cudaStatus;
}


hipError_t freeCudaMemory()
{
	hipError_t cudaStatus;

	// Free GPU buffers (faces, vertices, result, ...)   
	// Buffer for result picture

	// TODO Aufgabe 9
	cudaStatus = hipFree(result_Buffer);
	if (cudaStatus != hipSuccess) {
		const char *const err_str = hipGetErrorString(cudaStatus);
		std::cerr<<"freeCUDA for result_Buffer failed!!"<<std::endl;
		std::cerr << "Cuda error in " << __FILE__ << ":" << __LINE__ - 4
				  << ": " << err_str << " (" << cudaStatus << ")" << std::endl;
		return cudaStatus;
	}
	cudaStatus = hipFree(vertices_Buffer);
	if (cudaStatus != hipSuccess) {
		const char *const err_str = hipGetErrorString(cudaStatus);
		std::cerr<<"freeCUDA for vertices_Buffer failed!"<<std::endl;
		std::cerr << "Cuda error in " << __FILE__ << ":" << __LINE__ - 4
				  << ": " << err_str << " (" << cudaStatus << ")" << std::endl;
		return cudaStatus;
	}






	/////// DO NOT TOUCH FUNCTION CODE FROM HERE ON! ///////

	// Buffer for bounding spheres
	cudaStatus = hipFree(spheres_Buffer);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "freeCUDA for spheres_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for faces
	cudaStatus = hipFree(faces_Buffer);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "freeCUDA for faces_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for normals
	cudaStatus = hipFree(normals_Buffer);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "freeCUDA for normals_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for counts
	cudaStatus = hipFree(counts_Buffer);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "freeCUDA for counts_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for materials
	cudaStatus = hipFree(materials_Buffer);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "freeCUDA for materials_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for material indices
	cudaStatus = hipFree(matidx_Buffer);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "freeCUDA for matidx_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for object bounds
	cudaStatus = hipFree(objbounds_Buffer);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "freeCUDA for objbounds_Buffer failed!");
		return cudaStatus;
	}

	// Buffer for object meshes
	cudaStatus = hipFree(objmeshes_Buffer);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "freeCUDA for objmeshes_Buffer failed!");
		return cudaStatus;
	}

	return cudaStatus;
}


__device__ float4 normalizeDirection(float4 d)
{
    float4 nd;
    // TODO Aufgabe 6
	float base = sqrt(d.x*d.x+d.y*d.y+d.z*d.z);
	nd.x = d.x/base;
	nd.y = d.y/base;
	nd.z = d.z/base;
	nd.w = 0.0f;
    return nd;
}

__device__ float4 invertDirection(float4 d)
{
    float4 nd;
    nd.x = -d.x;
    nd.y = -d.y;
    nd.z = -d.z;
    return nd;
}

__device__ float distanceBetweenPoints(float4 p0, float4 p1)
{
    return		sqrtf((p0.x - p1.x) * (p0.x - p1.x) +
        (p0.y - p1.y) * (p0.y - p1.y) +
        (p0.z - p1.z) * (p0.z - p1.z));
}




__device__ float4 getCamRay3D(const float* cam, int x, int y, int2 resinfo)
{
    float4 d;
    d.x = x * cam[5] - 0.5f * resinfo.x * cam[5];
    d.y = y * cam[6] - 0.5f * resinfo.y * cam[6];
    d.z = -1.0f;
    d.w = 0.0f;
    d = normalizeDirection(d);
    return d;
}

__device__ float4 reflect3DLA(float4 l, float4 n)
{
    float4 r;
    // TODO Aufgabe 7
	r.w = 0.0f;
	float temp = 2*(l.x*n.x+l.y*n.y+l.z*n.z);
	r.x = temp*n.x-l.x;
	r.y = temp*n.y-l.y;
	r.z = temp*n.z-l.z;
    return r;
}

/*
Tests whether a 3D Ray hits a bounding Sphere
*/
__device__ bool rayIntersectsSphere(float4 s, float4 r, float4 o)
{
	// TODO Aufgabe 8
	float4 k;
	k.x = o.x - s.x;
	k.y = o.y - s.y;
	k.z = o.z - s.z;
	float beta = r.x*k.x+r.y*k.y + r.z*k.z;
	beta = beta*beta;
	float gama = (k.x*k.x+k.y*k.y+k.z*k.z)-s.w*s.w;
	if(beta>=gama)
		return true;
	else
		return false;
}

/*
Tests whether a 3D Ray hits a Triangle and returns barycentric coordinates of the intersection point
*/
__device__ bool rayIntersectsTriangle(float4 p0, float4 p1, float4 p2, float4 r, float4 o, float* u_t, float* v_t)
{
	float4 edge1;
	edge1.x = p1.x - p0.x;
	edge1.y = p1.y - p0.y;
	edge1.z = p1.z - p0.z;

	float4 edge2;
	edge2.x = p2.x - p0.x;
	edge2.y = p2.y - p0.y;
	edge2.z = p2.z - p0.z;

	// Direction pvec = dir*edge2;
	float4 pvec;
	pvec.x = r.y * edge2.z - r.z * edge2.y;
	pvec.y = r.z * edge2.x - r.x * edge2.z;
	pvec.z = r.x * edge2.y - r.y * edge2.x;

	//float det = edge1%pvec;
	float det = edge1.x * pvec.x + edge1.y * pvec.y + edge1.z * pvec.z;

	if (det > -2e-6f && det < 2e-6f)
		return false;

	float inv_det = 1.0f / det;

	// 	tvec = Direction(&orig, &p0);
	float4 tvec;
	tvec.x = o.x - p0.x;
	tvec.y = o.y - p0.y;
	tvec.z = o.z - p0.z;

	// float u = (tvec%pvec) * inv_det;
	float u = (tvec.x * pvec.x + tvec.y * pvec.y + tvec.z * pvec.z) * inv_det;
	if (u < 2e-6f || u > 1.00f + 2e-6f)
		return false;

	// qvec = tvec*edge1;	
	float4 qvec;
	qvec.x = tvec.y * edge1.z - tvec.z * edge1.y;
	qvec.y = tvec.z * edge1.x - tvec.x * edge1.z;
	qvec.z = tvec.x * edge1.y - tvec.y * edge1.x;

	// 	float v = (dir%qvec) * inv_det;
	float v = (r.x * qvec.x + r.y * qvec.y + r.z * qvec.z) * inv_det;
	if (v < 2e-6f || u + v > 1.0f + 2e-6f)
		return false;

	//float t = (edge2%qvec) * inv_det;
	float t = (edge2.x * qvec.x + edge2.y * qvec.y + edge2.z * qvec.z) * inv_det;
	if (t < 2e-1f)
		return false;

	*u_t = u;
	*v_t = v;
	return true;
}

__device__ void findIntersectionPoint(float4 p0, float4 p1, float4 p2,
	float4 n0, float4 n1, float4 n2,
	float u, float v,
	float4* sp, float4* n)
{
	// interpolate point coordinates
	(*sp).x = (1 - u - v) * p0.x + u * p1.x + v * p2.x;
	(*sp).y = (1 - u - v) * p0.y + u * p1.y + v * p2.y;
	(*sp).z = (1 - u - v) * p0.z + u * p1.z + v * p2.z;

	// interpolate normal in intersection point
	(*n).x = (1 - u - v) * n0.x + u * n1.x + v * n2.x;
	(*n).y = (1 - u - v) * n0.y + u * n1.y + v * n2.y;
	(*n).z = (1 - u - v) * n0.z + u * n1.z + v * n2.z;
}

__device__ bool traceRay(float4 r, float4 o, float4* sp, float4* n, int* objID,
	float4* spheres, int4* faces, float4* vertices, float4* normals,
	int* counts, const int meshcnt, bool find_ip)
{
	// Für jedes Netz Schnitttests durchführen
	bool intfound = false;
	float mindist = 30E30;
	*objID = -1;
	int absfcnt = 0;

	for (int j = 0; j < meshcnt; j++)
	{
		float4 bsphere = spheres[j];
		bool indi = rayIntersectsSphere(bsphere, r, o);
		if (indi)
		{
			float4 actIP;
			float4 actN;

			for (int i = 0; i < counts[j]; i++)
			{
				float u = 0.0f;
				float v = 0.0f;

				// Aktuelles Dreieck auslesen
				int4 t = faces[absfcnt + i];
				float4 p0 = vertices[t.x];
				float4 p1 = vertices[t.y];
				float4 p2 = vertices[t.z];

				float4 n0 = normals[t.x];
				float4 n1 = normals[t.y];
				float4 n2 = normals[t.z];

				//Schnitttest mit Dreieck durchführen
				if (!rayIntersectsTriangle(p0, p1, p2, r, o, &u, &v))
				{
					continue;
				}

				else
				{
					// Wenn Schnitt prüfen ob der näheste
					if (find_ip == false)
						return true;

					intfound = true;
					findIntersectionPoint(p0, p1, p2, n0, n1, n2, u, v, &actIP, &actN);
					float actdist = distanceBetweenPoints(actIP, o);

					if (actdist < mindist)
					{
						*objID = j;
						mindist = actdist;
						*sp = actIP;
						*n = actN;
					}
				}
			}
		}

		// Offsets für Knoten und Dreiecke anpassen
		absfcnt += counts[j];
	}

	if (!intfound)
		return false;

	return true;
}

__device__ float4 shadePoint(float4 col, float4 sp, float4 n, int objID, float4 eye_r,
	float4* spheres, int4* faces, float4* vertices, float4* normals,
	int* counts, const int meshcnt,
	const float* lightdat, float* materials, float4* r,
	int* matidx)
{
	// Material auslesen
	float mat[16] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0 };

	for (int i = 0; i < 16; i++) {
		mat[i] = materials[matidx[objID] * 16 + i];
	}

	//ambient
	col.x += lightdat_Buffer[3] * mat[3];
	col.y += lightdat_Buffer[4] * mat[4];
	col.z += lightdat_Buffer[5] * mat[5];

	n = normalizeDirection(n);

	// diffuse
	float4 light;
	light.x = lightdat_Buffer[0] - sp.x;
	light.y = lightdat_Buffer[1] - sp.y;
	light.z = lightdat_Buffer[2] - sp.z;
	light = normalizeDirection(light);

	// scalarproduct
	float NDotL = light.x * n.x + light.y * n.y + light.z * n.z;
	if (NDotL < 0.0f)
		NDotL = 0.0f;

	eye_r = invertDirection(eye_r);
	*r = reflect3DLA(eye_r, n);
	*r = normalizeDirection(*r);

	if (NDotL > 0.0f)
	{
		// Test for Shadowrays
		float4 tmp;
		int tmpi;
		if (traceRay(light, sp, &tmp, &tmp, &tmpi,
			spheres, faces, vertices, normals,
			counts, meshcnt, false))
			return col;

		col.x += NDotL * lightdat_Buffer[6] * mat[6];
		col.y += NDotL * lightdat_Buffer[7] * mat[7];
		col.z += NDotL * lightdat_Buffer[8] * mat[8];

		// spekluar
		float RDotE = (*r).x * light.x + (*r).y * light.y + (*r).z * light.z;
		//if (RDotE < 0.0f)
		//	RDotE = 0.0f;

		if (RDotE > 0.0f)
		{
			col.x += lightdat_Buffer[9] * mat[9] * powf(RDotE, mat[12]);
			col.y += lightdat_Buffer[10] * mat[10] * powf(RDotE, mat[12]);
			col.z += lightdat_Buffer[11] * mat[11] * powf(RDotE, mat[12]);
		}

	}

	return col;
}

__global__
void traceLA(float4* output,
	const    int2    resinfo,
	float4* spheres,
	int4* faces,
	float4* vertices,
	float4* normals,
	int* counts,
	const	int		meshcnt,
	float* materials,
	int* matidx,
	float4* objbounds,
	int* objmeshes,
	const		int			objcnt
)
{	
	// Locate the actual thread in the global scope

    // TODO Aufgabe 5):
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if(globalIdx >= resinfo.x) return;
	int globalIdy = blockIdx.y * blockDim.y + threadIdx.y;
    if(globalIdy >= resinfo.y) return;
	int globalIdSerial = globalIdy*resinfo.x+globalIdx;

//	// BLOCK TO TEST INDICES
//	float4 testcol = {0,0,0,0};
//	testcol.x = globalIdx / (float)(resinfo.x + 1);
//	testcol.y = globalIdy / (float)(resinfo.x + 1);
//	output[globalIdSerial] = testcol;
//	return;
	// BLOCK TO TEST INDICES

	// Sichtstrahl für Position erstellen
	float4 eye_r = getCamRay3D(camera_Buffer, globalIdx, globalIdy, resinfo);
	float4 campos;
	campos.x = camera_Buffer[0];
	campos.y = camera_Buffer[1];
	campos.z = camera_Buffer[2];

	// Farbe des Ausgabepixels
	float4 col;
	col.x = 0.0f;
	col.y = 0.0f;
	col.z = 0.0f;

	float4 sp; // Schnittpunkt mit der Dreiecksfläche
	float4 n; // Interpolierte Flächennormale
	int objID = -1;

	// Schnitttest
	if (!traceRay(eye_r, campos, &sp, &n, &objID,
		spheres, faces, vertices, normals,
		counts, meshcnt, true))
		return;

	// Farbe berechnen
	float4 rdir;
	col = shadePoint(col, sp, n, objID, eye_r,
		spheres, faces, vertices, normals,
		counts, meshcnt,
		0, materials, &rdir, matidx);

	// Farbwert in Ausgabe
	output[globalIdSerial] = col;

	// Reflektion in der ersten Stufe berechnen
	objID = -1;
	float4 nsp; float4 nn; float4 nrdir;

	if (!traceRay(rdir, sp, &nsp, &nn, &objID,
		spheres, faces, vertices, normals,
		counts, meshcnt, true))
		return;

	col = shadePoint(col, nsp, nn, objID, rdir,
		spheres, faces, vertices, normals,
		counts, meshcnt,
		0, materials, &nrdir, matidx);
	output[globalIdSerial] = col;

	return;
}

hipError_t traceWithCuda(float* result, const int blocks_x, const int blocks_y,
	SplitScene* scene,
	const int width, const int height, const int facecnt,
	float* spheres, int* faces, float* vertices, float* normals,
	int* loc_counts, float* materials, int* matidx,
	float* objbounds, int* objmeshes	
	) {
	hipError_t cudaStatus;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Initialize Device
	cudaStatus = setupCuda();
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}

	// Copy Data to Device Memory
	cudaStatus = setupCudaMemory(scene,
		width, height, facecnt,
		spheres, faces, vertices, normals,
		loc_counts, materials, matidx,
		objbounds, objmeshes);
	if (cudaStatus != hipSuccess) {
		return cudaStatus;
	}

	// TODO Aufgabe 10
	// Replace this default with Aufgabe 10:
//	dim3 threads(8, 8);
//	dim3 grid(width / threads.x, height / threads.y);
    int blockSize;
    int minGridSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, traceLA,0, height*width);
    cout<<"blocksize:"<<blockSize<<endl;
    cout<<"minGridSize: "<<minGridSize<<endl;
    int temp = sqrt(blockSize);
    dim3 threads(32,28);
    dim3 grid(width / threads.x, height / threads.y);


	cout << "Executing Kernel with grid ("
		<< grid.x << "," << grid.y << ")"
		<< " and block (" << threads.x << "," << threads.y << ")"
		<< endl;


	hipEventRecord(start,0);
	// Execute
	traceLA << < grid, threads >> > (result_Buffer,
		resinfo_Buffer,
		spheres_Buffer,
		faces_Buffer,
		vertices_Buffer,
		normals_Buffer,
		counts_Buffer,
		scene->getMeshCnt(),
		materials_Buffer,
		matidx_Buffer,
		objbounds_Buffer,
		objmeshes_Buffer,
		scene->objcnt
		);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "traceLA launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	std::cout << "GPU rendering required " << milliseconds/1000.0f << "s." << std::endl;

	// Copy output from GPU to host

    // TODO Aufgabe 4)
    cudaStatus = hipMemcpy(result,result_Buffer, width*height* sizeof(float)*4,hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
            const char *const err_str = hipGetErrorString(cudaStatus);
            std::cerr<<"result memory copy failed!"<<std::endl;
            std::cerr << "Cuda error in " << __FILE__ << ":" << __LINE__ - 4
                      << ": " << err_str << " (" << cudaStatus << ")" << std::endl;
		return cudaStatus;
    }

    cudaStatus = freeCudaMemory();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "freeCudaMemory failed!");
	}
	std::cout<<" finished"<<std::endl;

	return cudaStatus;
}