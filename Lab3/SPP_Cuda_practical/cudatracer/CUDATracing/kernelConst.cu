#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

/////////////// HOST FUNCTIONS ////////////////////////

hipError_t setupCuda() {
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	return cudaStatus;
}


/////////////// DEVICE FUNCTIONS ////////////////////////
__device__ float4 normalizeDirection(float4 d)
{
    float4 nd;
    float abs = sqrtf(d.x * d.x + d.y * d.y + d.z * d.z);
    nd.x = d.x / abs;
    nd.y = d.y / abs;
    nd.z = d.z / abs;
    return nd;
}

__device__ float4 invertDirection(float4 d)
{
    float4 nd;
    nd.x = -d.x;
    nd.y = -d.y;
    nd.z = -d.z;
    return nd;
}

__device__ float distanceBetweenPoints(float4 p0, float4 p1)
{
    return		sqrtf((p0.x - p1.x) * (p0.x - p1.x) +
        (p0.y - p1.y) * (p0.y - p1.y) +
        (p0.z - p1.z) * (p0.z - p1.z));
}

__device__ float4 getCamRay3D(const float* cam, int x, int y, int2 resinfo)
{
    float4 d;
    d.x = x * cam[5] - 0.5f * resinfo.x * cam[5];
    d.y = y * cam[6] - 0.5f * resinfo.y * cam[6];
    d.z = -1.0f;
    d.w = 0.0f;
    d = normalizeDirection(d);
    return d;
}

/*
Tests whether a 3D Ray hits a Triangle and returns barycentric coordinates of the intersection point
*/
__device__ bool rayIntersectsTriangle(float4 p0, float4 p1, float4 p2, float4 r, float4 o, float* u_t, float* v_t)
{
	float4 edge1;
	edge1.x = p1.x - p0.x;
	edge1.y = p1.y - p0.y;
	edge1.z = p1.z - p0.z;

	float4 edge2;
	edge2.x = p2.x - p0.x;
	edge2.y = p2.y - p0.y;
	edge2.z = p2.z - p0.z;

	// Direction pvec = dir*edge2;
	float4 pvec;
	pvec.x = r.y * edge2.z - r.z * edge2.y;
	pvec.y = r.z * edge2.x - r.x * edge2.z;
	pvec.z = r.x * edge2.y - r.y * edge2.x;

	//float det = edge1%pvec;
	float det = edge1.x * pvec.x + edge1.y * pvec.y + edge1.z * pvec.z;

	if (det > -2e-6f && det < 2e-6f)
		return false;

	float inv_det = 1.0f / det;

	// 	tvec = Direction(&orig, &p0);
	float4 tvec;
	tvec.x = o.x - p0.x;
	tvec.y = o.y - p0.y;
	tvec.z = o.z - p0.z;

	// float u = (tvec%pvec) * inv_det;
	float u = (tvec.x * pvec.x + tvec.y * pvec.y + tvec.z * pvec.z) * inv_det;
	if (u < 2e-6f || u > 1.00f + 2e-6f)
		return false;

	// qvec = tvec*edge1;	
	float4 qvec;
	qvec.x = tvec.y * edge1.z - tvec.z * edge1.y;
	qvec.y = tvec.z * edge1.x - tvec.x * edge1.z;
	qvec.z = tvec.x * edge1.y - tvec.y * edge1.x;

	// 	float v = (dir%qvec) * inv_det;
	float v = (r.x * qvec.x + r.y * qvec.y + r.z * qvec.z) * inv_det;
	if (v < 2e-6f || u + v > 1.0f + 2e-6f)
		return false;

	//float t = (edge2%qvec) * inv_det;
	float t = (edge2.x * qvec.x + edge2.y * qvec.y + edge2.z * qvec.z) * inv_det;
	if (t < 2e-1f)
		return false;

	*u_t = u;
	*v_t = v;
	return true;
}

__device__ void findIntersectionPoint(float4 p0, float4 p1, float4 p2,
	float4 n0, float4 n1, float4 n2,
	float u, float v,
	float4* sp, float4* n)
{
	// interpolate point coordinates
	(*sp).x = (1 - u - v) * p0.x + u * p1.x + v * p2.x;
	(*sp).y = (1 - u - v) * p0.y + u * p1.y + v * p2.y;
	(*sp).z = (1 - u - v) * p0.z + u * p1.z + v * p2.z;

	// interpolate normal in intersection point
	(*n).x = (1 - u - v) * n0.x + u * n1.x + v * n2.x;
	(*n).y = (1 - u - v) * n0.y + u * n1.y + v * n2.y;
	(*n).z = (1 - u - v) * n0.z + u * n1.z + v * n2.z;
}

__device__ bool traceRay(float4 r, float4 o, float4* sp, float4* n, int* objID,
	float4* spheres, int4* faces, float4* vertices, float4* normals,
	int* counts, const int meshcnt, bool find_ip)
{
	// Für jedes Netz Schnitttests durchführen
	bool intfound = false;
	float mindist = 30E30;
	*objID = -1;
	int absfcnt = 0;

	for (int j = 0; j < meshcnt; j++)
	{
		float4 bsphere = spheres[j];
		bool indi = rayIntersectsSphere(bsphere, r, o);
		if (indi)
		{
			float4 actIP;
			float4 actN;

			for (int i = 0; i < counts[j]; i++)
			{
				float u = 0.0f;
				float v = 0.0f;

				// Aktuelles Dreieck auslesen
				int4 t = faces[absfcnt + i];
				float4 p0 = vertices[t.x];
				float4 p1 = vertices[t.y];
				float4 p2 = vertices[t.z];

				float4 n0 = normals[t.x];
				float4 n1 = normals[t.y];
				float4 n2 = normals[t.z];

				//Schnitttest mit Dreieck durchführen
				if (!rayIntersectsTriangle(p0, p1, p2, r, o, &u, &v))
				{
					continue;
				}

				else
				{
					// Wenn Schnitt prüfen ob der näheste
					if (find_ip == false)
						return true;

					intfound = true;
					findIntersectionPoint(p0, p1, p2, n0, n1, n2, u, v, &actIP, &actN);
					float actdist = distanceBetweenPoints(actIP, o);

					if (actdist < mindist)
					{
						*objID = j;
						mindist = actdist;
						*sp = actIP;
						*n = actN;
					}
				}
			}
		}

		// Offsets für Knoten und Dreiecke anpassen
		absfcnt += counts[j];
	}

	if (!intfound)
		return false;

	return true;
}

